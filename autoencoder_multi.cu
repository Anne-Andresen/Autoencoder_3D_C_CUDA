#include "hip/hip_runtime.h"
#include <autoencoder.h>
#include <loss.h>
#include <utils.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <conv3d.h>

#define NUM_KERNELS = 3

typedef struct {
    Conv3D convs[NUM_KERNELS];
} ConvLayer;

typedef struct {
    ConvLayer conv1;
    ConvLayer conv2;
} Encoder;

typedef struct {
    ConvLayer deconv1;
    ConvLayer deconv2;
} Decoder;

typedef struct {
    Encoder encoder;
    Decoder decoder;
} Autoencoder;


#define cudaErrorCheck() {
    hipError_t e = hipGetLastError();
    if (e != hipSuccess) {
        print("CUDA error %s:%d: %s\n", __FILE__, __LINE__, cudaErrorString(e));
        exit(EXIT_FAILURE);
    }
}


void init_conv_layer(ConvLayer* layer, int inputDepth, int inputHeight, int inputWidth, int kernelSize) {
    for (int i = 0; i < NUM_KERNELS; i++) {
        conv3d_init(&layer->convs[i], inputDepth, inputHeight, inputWidth, kernelSize, kernelSize, kernelSize);
    }
}


void init_encoder(Encoder* encoder, int inputDepth, int inputHeight, int inputWidth, int kernelSize) {
    init_conv_layer(&encoder->conv1, inputDepth, inputHeight, inputWidth, kernelSize);
    init_conv_layer(&encoder->conv2, inputDepth, inputHeight, inputWidth, kernelSize);
}

void init_decoder(Decoder* decoder, int inputDepth, int inputHeight, int inputWidth, int kernelSize) {
    init_conv_layer(&decoder->deconv1, inputDepth. inputHeight, inputWidth, kernelSize);
    init_conv_layer(&decoder->deconv2, inputDepth, inputHeight, inputWidth, kernelSize);
}

void init_autoencoder(Autoencoder* autoencoder, int inputDepth, int, inputHeight, int inputWidth, int kernelSize) {
    init_encoder(&autoencoder->encoder, inputDepth, inputHeight, inputWidth, kernelSize);
    init_decoder(&autoencoder->decoder, inputDepth, inputHeight, inputWidth, kernelSize);
}

__global__ void conv3d_forward_kernel() {

}


__global__ void conv3d_backward_kernel() {
    
}

void forward_conv_layer(ConvLayer* layer, float* d_input, float* d_output) {
    float* d_temp_output;
    hipMalloc(&d_temp_output, layer->convs[0].D * layer->convs[0].H * layer->convs[0].W * sizeof(float));
    float* d_inter_output = d_input;
    for (int i = 0; i < NUM_KERNELS; i++) {
        conv3d_forward_kernel<<<  /* grid and block dimensions */, /* shared memory size */, /* stream */ >>(// kernel params

        );
        cudaErrorCheck();
        if (i < NUM_KERNELS - 1) {
            d_inter_output = d_temp_output;
        }
        else {
            hipMemcpy(d_output, d_temp_output, layer->convs[i].D * layer->convs[i].H * layer->convs[i].W * sizeof(float), hipMemcpyDeviceToDevice);
        }
    }
    hipFree(d_temp_output);
}

void forward_encoder(Encoder* encoder, float* d_input, float* d_output) {
    float* d_inter_output;
    hipMalloc(&d_inter_output, encoder->conv1.convs[0].D * encoder->conv1.convs[0].H * encoder->conv1.convs[0].W * sizeof(float));
    forward_conv_layer(&encoder->conv1, d_input, d_inter_output);
    forward_conv_layer(&encoder->conv2, d_inter_output, d_output);

    hipFree(d_inter_output); 
}
void forward_decoder(Decoder* decoder, float* d_input, float* d_output) {
    float* d_inter_output;
    hipMalloc(&d_inter_output, decoder->deconv1.convs[0].D * decoder->deconv1.convs[0].H * decoder->deconv1.convs[0].W * sizeof(float));
    forward_conv_layer(&decoder->deconv1, d_input, d_inter_output);
    forward_conv_layer(&decoder->deconv2, d_inter_output, d_output);

    hipFree(d_input);
}

void forward_autoencoder(Autoencoder* autoencoder, float* d_input, float* d_output) {
    float* d_latent_space;
    hipMalloc(&d_latent_space, autoencoder->encoder.conv2.convs[NUM_KERNELS - 1].D * autoencoder->encoder.conv2.convs[NUM_KERNELS - 1].H * autoencoder->encoder.conv2.convs[NUM_KERNELS - 1].W * sizeof(float));
    forward_encoder(&autoencoder->encoder, d_input, d_latent_space);
    forward_decoder(&autoencoder->decoder, d_latent_space, d_output);

    hipFree(d_latent_space);


    
}

void backward_conv_layer() {
    
}